#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include "simpleLinearBlurFilter.hpp"

// Device memory pointers
__device__ uchar *d_r, *d_g, *d_b;
__device__ uchar *d_r_out, *d_g_out, *d_b_out;

/*
 * CUDA Kernel Device code
 *
 */
__global__ void applySimpleLinearBlurFilter(uchar *r, uchar *g, uchar *b)
{
    // Calculate thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int num_image_pixels = d_rows * d_columns;
    
    // Shared memory for storing pixel values
    extern __shared__ uchar shared_mem[];
    uchar *shared_r = shared_mem;
    uchar *shared_g = &shared_mem[blockDim.x + 2];
    uchar *shared_b = &shared_mem[2 * (blockDim.x + 2)];
    
    if(threadId < num_image_pixels)
    {
        // Calculate row and column from thread ID
        int row = threadId / d_columns;
        int col = threadId % d_columns;
        
        // Load data into shared memory with padding for left and right neighbors
        int shared_idx = threadIdx.x + 1; // +1 for left padding
        
        // Load current pixel
        shared_r[shared_idx] = r[threadId];
        shared_g[shared_idx] = g[threadId];
        shared_b[shared_idx] = b[threadId];
        
        // Load left neighbor (handle edge case)
        if (threadIdx.x == 0) {
            if (col > 0) {
                shared_r[0] = r[threadId - 1];
                shared_g[0] = g[threadId - 1];
                shared_b[0] = b[threadId - 1];
            } else {
                // Left edge - use current pixel value
                shared_r[0] = r[threadId];
                shared_g[0] = g[threadId];
                shared_b[0] = b[threadId];
            }
        }
        
        // Load right neighbor (handle edge case)
        if (threadIdx.x == blockDim.x - 1) {
            if (col < d_columns - 1) {
                shared_r[shared_idx + 1] = r[threadId + 1];
                shared_g[shared_idx + 1] = g[threadId + 1];
                shared_b[shared_idx + 1] = b[threadId + 1];
            } else {
                // Right edge - use current pixel value
                shared_r[shared_idx + 1] = r[threadId];
                shared_g[shared_idx + 1] = g[threadId];
                shared_b[shared_idx + 1] = b[threadId];
            }
        }
        
        // Sync threads so that shared memory is fully loaded
        __syncthreads();
        
        // Apply simple 3-pixel wide linear blur filter
        // Average the current pixel with its left and right neighbors
        uchar blurred_r, blurred_g, blurred_b;
        
        if (col == 0) {
            // Left edge: average current and right pixel
            blurred_r = (shared_r[shared_idx] + shared_r[shared_idx + 1]) / 2;
            blurred_g = (shared_g[shared_idx] + shared_g[shared_idx + 1]) / 2;
            blurred_b = (shared_b[shared_idx] + shared_b[shared_idx + 1]) / 2;
        } else if (col == d_columns - 1) {
            // Right edge: average left and current pixel
            blurred_r = (shared_r[shared_idx - 1] + shared_r[shared_idx]) / 2;
            blurred_g = (shared_g[shared_idx - 1] + shared_g[shared_idx]) / 2;
            blurred_b = (shared_b[shared_idx - 1] + shared_b[shared_idx]) / 2;
        } else {
            // Middle pixels: average left, current, and right pixels
            blurred_r = (shared_r[shared_idx - 1] + shared_r[shared_idx] + shared_r[shared_idx + 1]) / 3;
            blurred_g = (shared_g[shared_idx - 1] + shared_g[shared_idx] + shared_g[shared_idx + 1]) / 3;
            blurred_b = (shared_b[shared_idx - 1] + shared_b[shared_idx] + shared_b[shared_idx + 1]) / 3;
        }
        
        // Sync threads before writing results
        __syncthreads();
        
        // Write blurred values back to global memory
        r[threadId] = blurred_r;
        g[threadId] = blurred_g;
        b[threadId] = blurred_b;
    }
}

__host__ float compareColorImages(uchar *r0, uchar *g0, uchar *b0, uchar *r1, uchar *g1, uchar *b1, int rows, int columns)
{
    cout << "Comparing actual and test pixel arrays\n";
    int numImagePixels = rows * columns;
    int imagePixelDifference = 0.0;

    for(int r = 0; r < rows; ++r)
    {
        for(int c = 0; c < columns; ++c)
        {
            // Fixed indexing bug: should be r*columns+c, not r*rows+c
            uchar image0R = r0[r*columns+c];
            uchar image0G = g0[r*columns+c];
            uchar image0B = b0[r*columns+c];
            uchar image1R = r1[r*columns+c];
            uchar image1G = g1[r*columns+c];
            uchar image1B = b1[r*columns+c];
            imagePixelDifference += ((abs(image0R - image1R) + abs(image0G - image1G) + abs(image0B - image1B))/3);
        }
    }

    float meanImagePixelDifference = imagePixelDifference / numImagePixels;
    float scaledMeanDifferencePercentage = (meanImagePixelDifference / 255);
    printf("meanImagePixelDifference: %f scaledMeanDifferencePercentage: %f\n", meanImagePixelDifference, scaledMeanDifferencePercentage);
    return scaledMeanDifferencePercentage;
}

__host__ void allocateDeviceMemory(int rows, int columns)
{
    //Allocate device constant symbols for rows and columns
    hipMemcpyToSymbol(HIP_SYMBOL(d_rows), &rows, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_columns), &columns, sizeof(int), 0, hipMemcpyHostToDevice);
}

__host__ void executeKernel(uchar *r, uchar *g, uchar *b, int rows, int columns, int threadsPerBlock)
{
    cout << "Executing kernel\n";
    //Launch the convert CUDA Kernel
    int blocksPerGrid = (rows * columns + threadsPerBlock - 1) / threadsPerBlock; // Ceiling division
    
    // Calculate shared memory size needed: 3 arrays * (threadsPerBlock + 2 for padding)
    size_t sharedMemSize = 3 * (threadsPerBlock + 2) * sizeof(uchar);

    applySimpleLinearBlurFilter<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(r, g, b);
    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    cout << "Cleaning CUDA device\n";
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ std::tuple<std::string, std::string, std::string, int> parseCommandLineArguments(int argc, char *argv[])
{
    cout << "Parsing CLI arguments\n";
    int threadsPerBlock = 256;
    std::string inputImage = "sloth.png";
    std::string outputImage = "grey-sloth.png";
    std::string currentPartId = "test";

    for (int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if (option.compare("-i") == 0)
        {
            inputImage = value;
        }
        else if (option.compare("-o") == 0)
        {
            outputImage = value;
        }
        else if (option.compare("-t") == 0)
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if (option.compare("-p") == 0)
        {
            currentPartId = value;
        }
    }
    cout << "inputImage: " << inputImage << " outputImage: " << outputImage << " currentPartId: " << currentPartId << " threadsPerBlock: " << threadsPerBlock << "\n";
    return {inputImage, outputImage, currentPartId, threadsPerBlock};
}

__host__ std::tuple<int, int, uchar *, uchar *, uchar *> readImageFromFile(std::string inputFile)
{
    cout << "Reading Image From File\n";
    Mat img = imread(inputFile, IMREAD_COLOR);
    
    const int rows = img.rows;
    const int columns = img.cols;
    size_t size = sizeof(uchar) * rows * columns;

    cout << "Rows: " << rows << " Columns: " << columns << "\n";

    uchar *r, *g, *b;
    hipMallocManaged(&r, size);
    hipMallocManaged(&g, size);
    hipMallocManaged(&b, size);
    
    for(int y = 0; y < rows; ++y)
    {
        for(int x = 0; x < columns; ++x)
        {
            Vec3b rgb = img.at<Vec3b>(y, x);
            // Fixed indexing: should be y*columns+x, not y*rows+x
            r[y*columns+x] = rgb.val[2]; // Red channel
            g[y*columns+x] = rgb.val[1]; // Green channel  
            b[y*columns+x] = rgb.val[0]; // Blue channel
        }
    }

    return {rows, columns, r, g, b};
}

__host__ std::tuple<uchar *, uchar *, uchar *>applyBlurKernel(std::string inputImage)
{
    cout << "CPU applying kernel\n";
    Mat img = imread(inputImage, IMREAD_COLOR);
    const int rows = img.rows;
    const int columns = img.cols;

    uchar *r = (uchar *)malloc(sizeof(uchar) * rows * columns);
    uchar *g = (uchar *)malloc(sizeof(uchar) * rows * columns);
    uchar *b = (uchar *)malloc(sizeof(uchar) * rows * columns);

    for(int y = 0; y < rows; ++y)
    {
        for(int x = 0; x < columns; ++x)
        {
            if (x == 0) {
                // Left edge: average current and right pixels
                Vec3b rgb1 = img.at<Vec3b>(y, x);
                Vec3b rgb2 = img.at<Vec3b>(y, x+1);
                b[y*columns+x] = (rgb1[0] + rgb2[0])/2;
                g[y*columns+x] = (rgb1[1] + rgb2[1])/2;
                r[y*columns+x] = (rgb1[2] + rgb2[2])/2;
            } else if (x == columns-1) {
                // Right edge: average left and current pixels
                Vec3b rgb0 = img.at<Vec3b>(y, x-1);
                Vec3b rgb1 = img.at<Vec3b>(y, x);
                b[y*columns+x] = (rgb0[0] + rgb1[0])/2;
                g[y*columns+x] = (rgb0[1] + rgb1[1])/2;
                r[y*columns+x] = (rgb0[2] + rgb1[2])/2;
            } else {
                // Middle pixels: average left, current, and right pixels
                Vec3b rgb0 = img.at<Vec3b>(y, x-1);
                Vec3b rgb1 = img.at<Vec3b>(y, x);
                Vec3b rgb2 = img.at<Vec3b>(y, x+1);
                b[y*columns+x] = (rgb0[0] + rgb1[0] + rgb2[0])/3;
                g[y*columns+x] = (rgb0[1] + rgb1[1] + rgb2[1])/3;
                r[y*columns+x] = (rgb0[2] + rgb1[2] + rgb2[2])/3;
            }
        }
    }

    return {r, g, b};
}

int main(int argc, char *argv[])
{
    std::tuple<std::string, std::string, std::string, int> parsedCommandLineArgsTuple = parseCommandLineArguments(argc, argv);
    std::string inputImage = get<0>(parsedCommandLineArgsTuple);
    std::string outputImage = get<1>(parsedCommandLineArgsTuple);
    std::string currentPartId = get<2>(parsedCommandLineArgsTuple);
    int threadsPerBlock = get<3>(parsedCommandLineArgsTuple);
    try 
    {
        auto[rows, columns, r, g, b] = readImageFromFile(inputImage);

        allocateDeviceMemory(rows, columns);
        executeKernel(r, g, b, rows, columns, threadsPerBlock);

        Mat colorImage(rows, columns, CV_8UC3);
        vector<int> compression_params;
        compression_params.push_back(IMWRITE_PNG_COMPRESSION);
        compression_params.push_back(9);

        for(int y = 0; y < rows; ++y)
        {
            for(int x = 0; x < columns; ++x)
            {
                // Fixed indexing: should be y*columns+x, not y*rows+x
                colorImage.at<Vec3b>(y,x) = Vec3b(b[y*columns+x], g[y*columns+x], r[y*columns+x]);
            }
        }

        imwrite(outputImage, colorImage, compression_params);

        auto[test_r, test_g, test_b] = applyBlurKernel(inputImage);
        
        float scaledMeanDifferencePercentage = compareColorImages(r, g, b, test_r, test_g, test_b, rows, columns) * 100;
        cout << "Mean difference percentage: " << scaledMeanDifferencePercentage << "\n";

        // Free managed memory
        hipFree(r);
        hipFree(g);
        hipFree(b);
        
        // Free CPU memory
        free(test_r);
        free(test_g);
        free(test_b);

        cleanUpDevice();
    }
    catch (Exception &error_)
    {
        cout << "Caught exception: " << error_.what() << endl;
        return 1;
    }
    return 0;
}